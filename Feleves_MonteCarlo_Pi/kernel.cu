#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <random>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include "..\Feleves_MonteCarlo_Pi\PreciseTimer.cpp"

#define ITERATION 100000
#define BLOCK_SIZE 1000
#define THREADS_PER_BLOCK 256

int points_within_circle = 0;
int points_within_square = 0;

double random_x, random_y, dist, pi;

double random_x_array[ITERATION];
double random_y_array[ITERATION];
int points_within_circle_array[ITERATION];
int points_within_square_array[ITERATION];
double pi_array[ITERATION];

//double test[10];

//__device__ double dev_test[10];
__device__ int dev_points_within_circle;
__device__ int dev_points_within_square;
__device__ double dev_random_x, dev_random_y, dev_dist, dev_pi;

__device__ double dev_random_x_array[ITERATION];
__device__ double dev_random_y_array[ITERATION];
__device__ int dev_points_within_circle_array[ITERATION];
__device__ int dev_points_within_square_array[ITERATION];
__device__ double dev_pi_array[ITERATION];

static std::default_random_engine rng = std::default_random_engine{};
static std::uniform_real_distribution<float> distribution(-1.0, 1.0);

void EstimatePi() {
	for (int i = 0; i < ITERATION; i++)
	{
		random_x = distribution(rng);

		random_y = distribution(rng);

		dist = random_x * random_x + random_y * random_y;

		if (dist <= 1)
			points_within_circle++;

		points_within_square++;

		pi = double(4 * points_within_circle) / points_within_square;

		std::cout << random_x << " " << random_y << " "
			<< points_within_circle << " " << points_within_square
			<< " - " << pi << std::endl
			<< std::endl;
	}
}

__global__ void EstimatePiSingle() {
	for (int i = 0; i < ITERATION; i++)
	{
		hiprandState state;

		hiprand_init(clock64(), i, 0, &state);

		dev_random_x = hiprand_uniform(&state);
		dev_random_y = hiprand_uniform(&state);

		dev_dist = dev_random_x * dev_random_x + dev_random_y * dev_random_y;

		if (dev_dist <= 1)
			dev_points_within_circle++;

		dev_points_within_square++;

		dev_pi = double(4 * dev_points_within_circle) / dev_points_within_square;

		dev_random_x_array[i] = dev_random_x;
		dev_random_y_array[i] = dev_random_y;
		dev_points_within_circle_array[i] = dev_points_within_circle;
		dev_points_within_square_array[i] = dev_points_within_square;
		dev_pi_array[i] = dev_pi;
	}
}

__global__ void EstimatePiN() {

	int i = threadIdx.x;
	hiprandState state;

	hiprand_init(clock64(), i, 0, &state);

	dev_random_x = hiprand_uniform(&state);
	dev_random_y = hiprand_uniform(&state);

	dev_dist = dev_random_x * dev_random_x + dev_random_y * dev_random_y;

	if (dev_dist <= 1)
		dev_points_within_circle++;

	dev_points_within_square++;

	dev_pi = double(4 * dev_points_within_circle) / dev_points_within_square;

	dev_random_x_array[i] = dev_random_x;
	dev_random_y_array[i] = dev_random_y;
	dev_points_within_circle_array[i] = dev_points_within_circle;
	dev_points_within_square_array[i] = dev_points_within_square;
	dev_pi_array[i] = dev_pi;
}

__global__ void EstimatePiNWithBlocks() {

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i >= ITERATION)
		return;

	hiprandState state;

	hiprand_init(clock64(), i, 0, &state);

	dev_random_x = hiprand_uniform(&state);
	dev_random_y = hiprand_uniform(&state);

	dev_dist = dev_random_x * dev_random_x + dev_random_y * dev_random_y;

	if (dev_dist <= 1)
		dev_points_within_circle++;  

	dev_points_within_square++;

	dev_pi = double(4 * dev_points_within_circle) / dev_points_within_square;

	dev_random_x_array[i] = dev_random_x;
	dev_random_y_array[i] = dev_random_y;
	dev_points_within_circle_array[i] = dev_points_within_circle;
	dev_points_within_square_array[i] = dev_points_within_square;
	dev_pi_array[i] = dev_pi;

}__global__ void EstimatePiShared() {
	__shared__ int shr_points_within_circle;
	__shared__ int shr_points_within_square;

	int i = blockIdx.x * blockDim.x + threadIdx.x;

	hiprandState state;

	hiprand_init(clock64(), i, 0, &state);

	dev_random_x = hiprand_uniform(&state);
	dev_random_y = hiprand_uniform(&state);

	dev_dist = dev_random_x * dev_random_x + dev_random_y * dev_random_y;

	if (dev_dist <= 1)
		dev_points_within_circle++;

	dev_points_within_square++;

	atomicAdd(&shr_points_within_circle, dev_points_within_circle);
	atomicAdd(&shr_points_within_square, dev_points_within_square);

	__syncthreads();

	if (i == 0)
	{
		atomicAdd(&dev_points_within_circle, shr_points_within_circle);
		atomicAdd(&dev_points_within_square, shr_points_within_square);

		dev_pi = double(4 * dev_points_within_circle) / dev_points_within_square;
	}

	dev_random_x_array[i] = dev_random_x;
	dev_random_y_array[i] = dev_random_y;
	dev_points_within_circle_array[i] = dev_points_within_circle;
	dev_points_within_square_array[i] = dev_points_within_square;
	dev_pi_array[i] = dev_pi;
}

__global__ void EstimatePiShared2() {
	__shared__ double shared_random_x_array[BLOCK_SIZE];
	__shared__ double shared_random_y_array[BLOCK_SIZE];

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i >= ITERATION)
		return;

	hiprandState state;

	hiprand_init(clock64(), i, 0, &state);

	shared_random_x_array[threadIdx.x] = hiprand_uniform(&state);
	shared_random_y_array[threadIdx.x] = hiprand_uniform(&state);

	__syncthreads();

	dev_random_x = shared_random_x_array[threadIdx.x];
	dev_random_y = shared_random_y_array[threadIdx.x];

	dev_dist = dev_random_x * dev_random_x + dev_random_y * dev_random_y;

	int dev_points_within_circle = 0;
	int dev_points_within_square = 0;

	if (dev_dist <= 1)
		dev_points_within_circle++;

	dev_points_within_square++;

	dev_pi = double(4 * dev_points_within_circle) / dev_points_within_square;

	dev_random_x_array[i] = dev_random_x;
	dev_random_y_array[i] = dev_random_y;
	dev_points_within_circle_array[i] = dev_points_within_circle;
	dev_points_within_square_array[i] = dev_points_within_square;
	dev_pi_array[i] = dev_pi;
}


void PrintOut() {
	for (int i = 0; i < ITERATION; i++)
	{
		std::cout << random_x_array[i] << " " << random_y_array[i] << " "
			<< points_within_circle_array[i] << " " << points_within_square_array[i]
			<< " - " << pi_array[i] << std::endl
			<< std::endl;
	}
}


int main()
{
	#pragma region CPU
	CPreciseTimer timer;
	timer.StartTimer();
	//EstimatePi();
	//timer.StopTimer();
	//float elapsedTime = timer.GetTimeSec();
	//std::cout << elapsedTime << std::endl;

	#pragma endregion

	#pragma region Single Thread
	hipMemcpyToSymbol(HIP_SYMBOL(dev_points_within_circle), &points_within_circle, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_points_within_square), &points_within_square, sizeof(int));

	/*EstimatePiSingle << < 1, 1 >> > ();
	

	hipMemcpyFromSymbol(random_x_array, HIP_SYMBOL(dev_random_x_array), ITERATION * sizeof(double));
	hipMemcpyFromSymbol(random_y_array, HIP_SYMBOL(dev_random_y_array), ITERATION * sizeof(double));
	hipMemcpyFromSymbol(points_within_circle_array, HIP_SYMBOL(dev_points_within_circle_array), ITERATION * sizeof(int));
	hipMemcpyFromSymbol(points_within_square_array, HIP_SYMBOL(dev_points_within_square_array), ITERATION * sizeof(int));
	hipMemcpyFromSymbol(pi_array, HIP_SYMBOL(dev_pi_array), ITERATION * sizeof(double));

	timer.StopTimer();

	PrintOut();
	float elapsedTime = timer.GetTimeSec();
	std::cout << elapsedTime << std::endl;*/

	#pragma endregion
	
	#pragma region N Threads

	//EstimatePiN << < 1, ITERATION >> > ();

	//hipMemcpyFromSymbol(random_x_array, HIP_SYMBOL(dev_random_x_array), ITERATION * sizeof(double));
	//hipMemcpyFromSymbol(random_y_array, HIP_SYMBOL(dev_random_y_array), ITERATION * sizeof(double));
	//hipMemcpyFromSymbol(points_within_circle_array, HIP_SYMBOL(dev_points_within_circle_array), ITERATION * sizeof(int));
	//hipMemcpyFromSymbol(points_within_square_array, HIP_SYMBOL(dev_points_within_square_array), ITERATION * sizeof(int));
	//hipMemcpyFromSymbol(pi_array, HIP_SYMBOL(dev_pi_array), ITERATION * sizeof(double));

	//timer.StopTimer();
	////PrintOut();
	//float elapsedTime = timer.GetTimeSec();
	//std::cout << elapsedTime << std::endl;

	#pragma endregion

	#pragma region N Threads With Blocks

	//int block_count = (ITERATION - 1) / BLOCK_SIZE + 1;
	//EstimatePiNWithBlocks << < block_count, BLOCK_SIZE >> > ();

	//hipMemcpyFromSymbol(random_x_array, HIP_SYMBOL(dev_random_x_array), ITERATION * sizeof(double));
	//hipMemcpyFromSymbol(random_y_array, HIP_SYMBOL(dev_random_y_array), ITERATION * sizeof(double));
	//hipMemcpyFromSymbol(points_within_circle_array, HIP_SYMBOL(dev_points_within_circle_array), ITERATION * sizeof(int));
	//hipMemcpyFromSymbol(points_within_square_array, HIP_SYMBOL(dev_points_within_square_array), ITERATION * sizeof(int));
	//hipMemcpyFromSymbol(pi_array, HIP_SYMBOL(dev_pi_array), ITERATION * sizeof(double));

	//timer.StopTimer();

	////PrintOut();
	//float elapsedTime = timer.GetTimeSec();
	//std::cout << elapsedTime << std::endl;

	#pragma endregion
	
	#pragma region Shared Memory

	int block_count = (ITERATION - 1) / BLOCK_SIZE + 1;
	EstimatePiShared << < block_count, BLOCK_SIZE >> > ();
	hipMemcpyFromSymbol(random_x_array, HIP_SYMBOL(dev_random_x_array), ITERATION * sizeof(double));
	hipMemcpyFromSymbol(random_y_array, HIP_SYMBOL(dev_random_y_array), ITERATION * sizeof(double));
	hipMemcpyFromSymbol(points_within_circle_array, HIP_SYMBOL(dev_points_within_circle_array), ITERATION * sizeof(int));
	hipMemcpyFromSymbol(points_within_square_array, HIP_SYMBOL(dev_points_within_square_array), ITERATION * sizeof(int));
	hipMemcpyFromSymbol(pi_array, HIP_SYMBOL(dev_pi_array), ITERATION * sizeof(double));

	timer.StopTimer();

	//PrintOut();
	float elapsedTime = timer.GetTimeSec();
	std::cout << elapsedTime << std::endl;

	#pragma endregion

	#pragma region Shared Memory 2

	//int block_count = (ITERATION - 1) / BLOCK_SIZE + 1;
	//EstimatePiShared2 << < block_count, BLOCK_SIZE >> > ();
	//hipMemcpyFromSymbol(random_x_array, HIP_SYMBOL(dev_random_x_array), ITERATION * sizeof(double));
	//hipMemcpyFromSymbol(random_y_array, HIP_SYMBOL(dev_random_y_array), ITERATION * sizeof(double));
	//hipMemcpyFromSymbol(points_within_circle_array, HIP_SYMBOL(dev_points_within_circle_array), ITERATION * sizeof(int));
	//hipMemcpyFromSymbol(points_within_square_array, HIP_SYMBOL(dev_points_within_square_array), ITERATION * sizeof(int));
	//hipMemcpyFromSymbol(pi_array, HIP_SYMBOL(dev_pi_array), ITERATION * sizeof(double));

	//timer.StopTimer();

	//PrintOut();
	//float elapsedTime = timer.GetTimeSec();
	//std::cout << elapsedTime << std::endl;

	#pragma endregion
}

